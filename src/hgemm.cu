#include <iostream>
#include <fstream>
#include <sstream>
#include <vector>
#include <cmath>
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <cassert>
#include <chrono>
#include <hip/hip_fp16.h>
#include <mma.h>

using namespace nvcuda;

bool read_matrices_from_dir(const std::string& dir,
                            std::vector<__half>& A_fp16,
                            std::vector<__half>& B_fp16,
                            int& M, int& N, int& K) {
    std::string path_A = dir + "/A_matrix.bin";
    std::string path_B = dir + "/B_matrix.bin";

    std::ifstream fa(path_A, std::ios::binary);
    std::ifstream fb(path_B, std::ios::binary);
    if (!fa.is_open() || !fb.is_open()) {
        std::cerr << "Error opening binary matrix files in " << dir << std::endl;
        return false;
    }

    int m_a = 0, k_a = 0, k_b = 0, n_b = 0;

    fa.read(reinterpret_cast<char*>(&m_a), sizeof(int));
    fa.read(reinterpret_cast<char*>(&k_a), sizeof(int));
    size_t size_A = static_cast<size_t>(m_a) * k_a;
    A_fp16.resize(size_A);
    fa.read(reinterpret_cast<char*>(A_fp16.data()), size_A * sizeof(__half));

    fb.read(reinterpret_cast<char*>(&k_b), sizeof(int));
    fb.read(reinterpret_cast<char*>(&n_b), sizeof(int));
    size_t size_B = static_cast<size_t>(k_b) * n_b;
    B_fp16.resize(size_B);
    fb.read(reinterpret_cast<char*>(B_fp16.data()), size_B * sizeof(__half));

    fa.close();
    fb.close();

    if (k_a != k_b) {
        std::cerr << "Error: K dimension mismatch between A and B\n";
        return false;
    }

    M = m_a;
    K = k_a;
    N = n_b;
    return true;
}

__global__ void wmma_gemm_kernel_fp16(const __half* A, const __half* B, __half* C, int M, int N, int K) {
    const int WMMA_M = 16;
    const int WMMA_N = 16;
    const int WMMA_K = 16;
    
    int M_pad = (M + WMMA_M - 1) / WMMA_M * WMMA_M;
    int N_pad = (N + WMMA_N - 1) / WMMA_N * WMMA_N;
    int K_pad = (K + WMMA_K - 1) / WMMA_K * WMMA_K;
    
    int warpM = (blockIdx.x * blockDim.x + threadIdx.x) / warpSize;
    int warpN = (blockIdx.y * blockDim.y + threadIdx.y);
    
    wmma::fragment<wmma::matrix_a, WMMA_M, WMMA_N, WMMA_K, __half, wmma::row_major> a_frag;
    wmma::fragment<wmma::matrix_b, WMMA_M, WMMA_N, WMMA_K, __half, wmma::row_major> b_frag;
    wmma::fragment<wmma::accumulator, WMMA_M, WMMA_N, WMMA_K, __half> c_frag;
    
    wmma::fill_fragment(c_frag, __float2half(0.0f));
    
    for (int k = 0; k < K_pad; k += WMMA_K) {
        if (warpM * WMMA_M < M && k + WMMA_K <= K) {
            wmma::load_matrix_sync(a_frag, A + warpM * WMMA_M * K + k, K);
        } else {
            for (int i = 0; i < a_frag.num_elements; i++) {
                a_frag.x[i] = __float2half(0.0f);
            }
        }
        
        if (warpN * WMMA_N < N && k + WMMA_K <= K) {
            wmma::load_matrix_sync(b_frag, B + k * N + warpN * WMMA_N, N);
        } else {
            for (int i = 0; i < b_frag.num_elements; i++) {
                b_frag.x[i] = __float2half(0.0f);
            }
        }
        
        wmma::mma_sync(c_frag, a_frag, b_frag, c_frag);
    }
    
    if (warpM * WMMA_M < M && warpN * WMMA_N < N) {
        wmma::store_matrix_sync(C + warpM * WMMA_M * N + warpN * WMMA_N, c_frag, N, wmma::mem_row_major);
    }
}

int main(int argc, char* argv[]) {
    std::string input_dir = "data/input/Case1_768x768x768";
    std::string output_dir = "data/output";

    for (int i = 1; i < argc; ++i) {
        std::string arg = argv[i];
        if ((arg == "-d" || arg == "--indir") && i + 1 < argc) {
            input_dir = argv[++i];
        } else if ((arg == "-o" || arg == "--outdir") && i + 1 < argc) {
            output_dir = argv[++i];
        } else {
            std::cerr << "Unknown or incomplete argument: " << arg << std::endl;
            std::cerr << "Usage: " << argv[0] << " [-d input_dir] [-o output_dir]" << std::endl;
            return 1;
        }
    }

    std::string case_name = input_dir.substr(input_dir.find_last_of("/\\") + 1);
    std::string output_file = output_dir + "/result_" + case_name + ".txt";

    int M, N, K;
    std::vector<__half> A_fp16, B_fp16;
    if (!read_matrices_from_dir(input_dir, A_fp16, B_fp16, M, N, K)) return 1;

    __half *d_A_fp16, *d_B_fp16, *d_C_custom;
    hipMalloc(&d_A_fp16, M * K * sizeof(__half));
    hipMalloc(&d_B_fp16, K * N * sizeof(__half));
    hipMalloc(&d_C_custom, M * N * sizeof(__half));

    hipMemcpy(d_A_fp16, A_fp16.data(), M * K * sizeof(__half), hipMemcpyHostToDevice);
    hipMemcpy(d_B_fp16, B_fp16.data(), K * N * sizeof(__half), hipMemcpyHostToDevice);
    hipMemset(d_C_custom, 0, M * N * sizeof(__half));
    hipDeviceSynchronize();

    dim3 block(32, 4);
    dim3 grid((M + 15) / 16, (N + 15) / 16);

    auto start2 = std::chrono::high_resolution_clock::now();
    wmma_gemm_kernel_fp16<<<grid, block>>>(d_A_fp16, d_B_fp16, d_C_custom, M, N, K);
    hipDeviceSynchronize();
    auto end2 = std::chrono::high_resolution_clock::now();

    double duration_custom = std::chrono::duration<double, std::milli>(end2 - start2).count();

    std::vector<__half> C_custom_host(M * N);
    hipMemcpy(C_custom_host.data(), d_C_custom, M * N * sizeof(__half), hipMemcpyDeviceToHost);

    float sum_custom = 0.f;
    for (int i = 0; i < M * N; ++i) {
        sum_custom += __half2float(C_custom_host[i]);
    }

    double flops = 2.0 * M * N * K;
    double custom_gflops = flops / (duration_custom / 1000.0) / 1e9;

    std::cout << "WMMA FP16 Kernel Time: " << duration_custom << " ms, gFLOPS: " << custom_gflops << std::endl;
    std::cout << "WMMA Kernel Result sum: " << sum_custom << std::endl;

    std::ofstream out(output_file);
    if (out.is_open()) {
        out << "Case: " << case_name << "\n";
        out << "WMMA FP16 Kernel Time: " << duration_custom << " ms, gFLOPS: " << custom_gflops << "\n";
        out << "WMMA Kernel Result sum: " << sum_custom << "\n";
        out.close();
    }

    hipFree(d_A_fp16);
    hipFree(d_B_fp16);
    hipFree(d_C_custom);

    return 0;
}
